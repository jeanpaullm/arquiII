#include <stdio.h>
#include <hip/hip_runtime.h>

int *a, *b;  // host data
int *c, *c2;  // results

//Cuda error checking - non mandatory
void cudaCheckError() {
 hipError_t e=hipGetLastError();
 if(e!=hipSuccess) {
   printf("Cuda failure %s:%d: '%s'\n",__FILE__,__LINE__,hipGetErrorString(e));
   exit(0);
 }
}

//GPU kernel
__global__
void matrixMultiply(int* A,int* B,int* C, int N){
    int ai = threadIdx.y * blockDim.x;
    int bi = threadIdx.x;
    int ci = threadIdx.x + threadIdx.y * blockDim.x ;
    int result = 0;
    if(ci < N) {
      for(int i = 0; i < 4; i++){
          result += A[ai] * B[bi];
          ai++;
          bi += 4;
      }
      C[ci] = result;
    }
}

int main(int argc,char **argv) {

    printf("Begin \n");
    //Iterations
    int n=16; //cantidad de elementos de la matriz, x * y
    //Number of blocks
    int nBytes = n*sizeof(int);

    //memory allocation
    a = (int *) malloc(nBytes);
    b = (int *) malloc(nBytes);
    c = (int *) malloc(nBytes);
    c2 = (int *) malloc(nBytes);

    // Data filling
    for(int i=0;i<n;i++)
    a[i]=i,b[i]=i;

    printf("Allocating device memory on host..\n");
   //GPU memory allocation
    int *a_d,*b_d,*c_d;
    hipMalloc((void **) &a_d, n*sizeof(int));
    hipMalloc((void **) &b_d, n*sizeof(int));
    hipMalloc((void **) &c_d, n*sizeof(int));

    printf("Copying to device..\n");
    hipMemcpy(a_d, a, n*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(b_d, b, n*sizeof(int), hipMemcpyHostToDevice);

    //Work definition
    dim3 threadsPerBlock(4, 4);
    dim3 numBlocks(1, 1);

    printf("Doing GPU Vector Multiplication\n");
    matrixMultiply<<<numBlocks, threadsPerBlock>>>(a_d, b_d, c_d, n);
    cudaCheckError();

    //Wait for kernel call to finish
    hipDeviceSynchronize();

    //Copying data back to host, this is a blocking call and will not start until all kernels are finished
    printf("Copying back to host...\n");
    hipMemcpy(c, c_d, n*sizeof(int), hipMemcpyDeviceToHost);
    int counter = 0;
    
    printf("A\n");    
    for(int i = 0; i < 4; i++) {
	for(int j = 0; j < 4; j++) {
          printf("%d ",a[counter]);
	  counter++;
	}
	printf("\n");
    }

    counter = 0;

    printf("B\n");    
    for(int i = 0; i < 4; i++) {
	for(int j = 0; j < 4; j++) {
          printf("%d ",b[counter]);
	  counter++;
	}
	printf("\n");
    }
 
    counter = 0;

    printf("C\n");    
    for(int i = 0; i < 4; i++) {
	for(int j = 0; j < 4; j++) {
          printf("%d ",c[counter]);
	  counter++;
	}
	printf("\n");
    }

    //Free GPU memory
    hipFree(a_d);
    hipFree(b_d);
    hipFree(c_d);
    return 0;
}
