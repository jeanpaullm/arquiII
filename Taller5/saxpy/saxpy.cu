#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>

int *a, *b;  // host data
int *c, *c2;  // results

//Cuda error checking - non mandatory
void cudaCheckError() {
 hipError_t e=hipGetLastError();
 if(e!=hipSuccess) {
   printf("Cuda failure %s:%d: '%s'\n",__FILE__,__LINE__,hipGetErrorString(e));
   exit(0); 
 }
}

//GPU kernel 
__global__
void saxpy(int *A,int *B,int *C,int N){
    int ai = threadIdx.y * blockDim.x;
    int bi = threadIdx.x;
    int ci = threadIdx.x + threadIdx.y * blockDim.x ;
    int result = 0;
    if(ci < N*N) {
      for(int i = 0; i < N; i++) {
          result += A[ai] * B[bi];
          ai++;
          bi += N;
      }
      C[ci] = result + C[ci];
    } 
}

//CPU function, N matrix width
void saxpy_h(int *A1,int *B1, int *C1, int N){
    int ai, bi, result;
    for(int i = 0; i < N*N; i++) {
	ai = (i - i%N)/N; 
        bi = i%N;
        result = 0;
	for(int j = 0; j < N; j++) {
	    result = A1[ai] * B1[bi];
            ai++;
	    bi += N;
        }      
        C1[i] = result + C1[i];
    }
}

int main(int argc,char **argv) {
    printf("Begin \n");
    //Matrix side size 
    int n = 2500;

    //memory allocation	
    a = (int *) malloc(n*n*sizeof(int));
    b = (int *) malloc(n*n*sizeof(int));
    c = (int *) malloc(n*n*sizeof(int));
    c2 = (int *) malloc(n*n*sizeof(int));

    int *a_d,*b_d,*c_d;
   
    // Data filling
    for(int i=0; i<n*n; i++)
    a[i]=i,b[i]=i,c[i];


    printf("Allocating device memory on host..\n");
   //GPU memory allocation
    hipMalloc((void **) &a_d, n*n*sizeof(int));
    hipMalloc((void **) &b_d, n*n*sizeof(int));
    hipMalloc((void **) &c_d, n*n*sizeof(int));

    printf("Copying to device..\n");
    hipMemcpy(a_d, a, n*n*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(b_d, b, n*n*sizeof(int), hipMemcpyHostToDevice);

    dim3 threadsPerBlock(10,10);
    dim3 numBlocks((n*n)/(10*10),1);

    clock_t start_d=clock();
    printf("Doing GPU matrix saxpy\n");
    saxpy<<<numBlocks, threadsPerBlock>>>(a_d, b_d, c_d, n);
    cudaCheckError();

    //Wait for kernel call to finish
    hipDeviceSynchronize();

    clock_t end_d = clock();
    

    printf("Doing CPU Vector add\n");
    clock_t start_h = clock();
    saxpy_h(a, b, c2, n);
    clock_t end_h = clock();
	
    //Time computing
    double time_d = (double)(end_d-start_d)/CLOCKS_PER_SEC;
    double time_h = (double)(end_h-start_h)/CLOCKS_PER_SEC;

    //Copying data back to host, this is a blocking call and will not start until all kernels are finished
    hipMemcpy(c, c_d, n*n*sizeof(int), hipMemcpyDeviceToHost);
    printf("n = %d \t GPU time = %fs \t CPU time = %fs\n", n, time_d, time_h);

    //Free GPU memory
    hipFree(a_d);
    hipFree(b_d);
    hipFree(c_d);
    return 0;
}
